
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu(void) {
    printf("hello friends, from thread [%d, %d] \ From device \n", threadIdx.x,blockIdx.x);
}

int main(void) {
    printf("hello friends this is the host!\n");
    print_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}